/* 159.735 Semester 2, 2016.  Ian Bond, 3/10/2016
 
 Sequential version of the N-sphere counting problem for Assignment
 5. Two alternative algorithms are presented.

 Note: a rethink will be needed when implementing a GPU version of
 this. You can't just cut and paste code.

 To compile: g++ -O3 -o nsphere nsphere.cpp
 (you will get slightly better performance with the O3 optimization flag)
*/
#include <cstdlib>
#include <cmath>

#include <iostream>
#include <string>

#include <vector>
#include <hip/hip_runtime.h>

const long MAXDIM = 10;
const double RMIN = 2.0;
const double RMAX = 8.0;

long powlong(long n, long k)
/* Evaluate n**k where both are long integers */
{
  long p = 1;
  for (long i = 0; i < k; ++i) p *= n;
  return p;
}

/*----------------------------------------------------------------------------*/


__global__ void count_in_v1_gpu (long base, long halfb, double rsquare, long ndim , float* count )
{
  int n = blockDim.x * blockIdx.x + threadIdx.x;

  
  // Indices in x,y,z,.... 

  long* index = (long*)malloc(ndim * sizeof(long));


  for (long i = 0; i < ndim; ++i) index[i] = 0;
  
  long idx = 0;
  while (n != 0) {
    long rem = n % base;
    n = n / base;
    index[idx] = rem;
    ++idx;
  }

  double rtestsq = 0;

  for (long k = 0; k < ndim; ++k) {
    double xk = index[k] - halfb;
    rtestsq += xk * xk;
  }
  if (rtestsq < rsquare) atomicAdd(count,1.0);

}



void convert(long num, long base, std::vector<long>& index)
/* Convert a decimal number into another base system - the individual
   digits in the new base are stored in the index array. */
{
  const long ndim = index.size();
  for (long i = 0; i < ndim; ++i) index[i] = 0;
  long idx = 0;
  while (num != 0) {
    long rem = num % base;
    num = num / base;
    index[idx] = rem;
    ++idx;
  }
}

long count_in_v1(long ndim, double radius)
/* 
   Version 1 of the counting algorithm. Given:

   ndim   -> number of dimensions of the hypersphere
   radius -> radius of the hypersphere

   count the number of integer points that lie wholly within the
   hypersphere, assuming it is centred on the origin.
*/
{
  const long halfb = static_cast<long>(floor(radius));
  const long base = 2 * halfb + 1;
  const double rsquare = radius * radius;

  // This is the total number of points we will need to test.
  const long ntotal = powlong(base, ndim);

  long count = 0;

  // Indices in x,y,z,.... 
  std::vector<long> index(ndim, 0);

  // Loop over the total number of points. For each visit of the loop,
  // we covert n to its equivalent in a number system of given "base".
  for (long n = 0; n < ntotal; ++n) {
    convert(n, base, index);
    double rtestsq = 0;
    for (long k = 0; k < ndim; ++k) {
      double xk = index[k] - halfb;
      rtestsq += xk * xk;
    }
    if (rtestsq < rsquare) ++count;
  }

  return count;
}

/*----------------------------------------------------------------------------*/

void addone(std::vector<long>& index, long base, long i)
/* Add one to a digital counter of given base. When one digit hits
   maximum, it is necessary to carry one over into the next
   column. This is done recursively here. */
{
  long ndim = index.size();
  long newv = index[i] + 1;
  if (newv >= base) {
    index[i] = 0;
    if (i < ndim - 1) addone(index, base, i+1);
  }
  else {
    index[i] = newv;
  }
}

long count_in_v2(long ndim, double radius)
/* 
   Version 2 of the counting algorithm. Given:

   ndim   -> number of dimensions of the hypersphere
   radius -> radius of the hypersphere

   count the number of integer points that lie wholly within the
   hypersphere, assuming it is centred on the origin.
*/
{
  const long halfb = static_cast<long>(floor(radius));
  const long base = 2 * halfb + 1;
  const double rsquare = radius * radius;
  const long ntotal = powlong(base, ndim);

  long count = 0;

  // This is the counter
  std::vector<long> index(ndim, 0);

  // Loop over the total number of points to test, ticking over the
  // counter as we go.
  for (long n = 0; n < ntotal; ++n) {
    double rtestsq = 0;
    for (long k = 0; k < ndim; ++k) {
      double xk = index[k] - halfb;
      rtestsq += xk * xk;
    }
    if (rtestsq < rsquare) ++count;
    addone(index, base, 0);
  }
  return count;
}


int main(int argc, char* argv[]) 
{
  // You can make this larger if you want
  const long ntrials = 20;


  for (long n = 0; n < ntrials; ++n) {

    
    // Get a random value for the hypersphere radius between the two limits
    const double r = drand48() * (RMAX - RMIN) + RMIN;

    // Get a random value for the number of dimensions between 1 and
    // MAXDIM inclusive
    const long  nd = lrand48() % (MAXDIM - 1) + 1;
    std::cout << "### " << n << " " << r << " " << nd << " ... " << std::endl;

    const long halfb = static_cast<long>(floor(r));
    const long base = 2 * halfb + 1;
    const long ntotal = powlong(base, nd);
    const double rsquare = r * r;

    float* h_count = (float*)malloc(sizeof(float));
    float *d_count;

    hipMalloc(&d_count, sizeof(float));
    int threadsPerBlock = 256;
    int blocksPerGrid = ntotal / threadsPerBlock;

    count_in_v1_gpu<<<blocksPerGrid, threadsPerBlock>>>( base, halfb, rsquare, nd, d_count );
    hipMemcpy( h_count, d_count, sizeof(float), hipMemcpyDeviceToHost);

    
    //const long num1 = count_in_v1(nd, r);
    //const long num2 = count_in_v2(nd, r);
    std::cout << " -> " << h_count[0] << std::endl;
    hipFree(d_count);
  }






}

