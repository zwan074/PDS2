/* 159.735 Semester 2, 2016.  Ian Bond, 3/10/2016
 
 Sequential version of the N-sphere counting problem for Assignment
 5. Two alternative algorithms are presented.

 Note: a rethink will be needed when implementing a GPU version of
 this. You can't just cut and paste code.

 To compile: g++ -O3 -o nsphere nsphere.cpp
 (you will get slightly better performance with the O3 optimization flag)
*/
#include <cstdlib>
#include <cmath>

#include <iostream>
#include <string>

#include <vector>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void inc(unsigned long long int *foo) {
  atomicAdd(foo, 1);
}

int main() {
  unsigned long long int count = 0, *cuda_count;
  hipMalloc((void**)&cuda_count, sizeof(unsigned long long int));
  hipMemcpy(cuda_count, &count, sizeof(unsigned long long int), hipMemcpyHostToDevice);
  cout << "count: " << count << '\n';
  inc <<< 100, 25 >>> (cuda_count);
  hipMemcpy(&count, cuda_count, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
  hipFree(cuda_count);
  cout << "count: " << count << '\n';
  return 0;
}
