/* Vector addition deom on GPU

   To compile: nvcc -o testprog1 testprog1.cu

 */
#include <iostream>

#include <hip/hip_runtime.h>

// Kernel that executes on the CUDA device. This is executed by ONE
// stream processor
__global__ void vec_add(float* A, float* B, float* C, int N)
{
  // What element of the array does this thread work on
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N) 
    C[i] = A[i] + B[i];
}

// main routine that executes on the host
int main(void)
{
  int n;
  int N = 10000000;
  size_t size = N * sizeof(float);

  // Allocate in HOST memory
  float* h_A = (float*)malloc(size);
  float* h_B = (float*)malloc(size);
  float* h_C = (float*)malloc(size);

  // Initialize vectors
  for (n = 0; n < N; ++n) {
    h_A[n] = 3.2333 * n;
    h_B[n] = 8.09287 * n;
  }

  // Allocate in DEVICE memory
  float *d_A, *d_B, *d_C;
  hipMalloc(&d_A, size);
  hipMalloc(&d_B, size);
  hipMalloc(&d_C, size);

  // Copy vectors from host to device memory
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  // Invoke kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  std::cout << "Launching a grid of " 
	    << blocksPerGrid << " "
	    << threadsPerBlock * blocksPerGrid
	    << " threads" << std::endl;
  vec_add<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

  // Copy result from device memory into host memory
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  // Print the first and last 10 elements of the arrays
  for (n = 0; n < N; ++n) {
    if (n < 10 || n >= N - 10) 
      std::cout << n << " " << h_A[n] << " " << h_B[n] 
		<< " " << h_C[n] << std::endl;
  } 

  free(h_A);
  free(h_B);
  free(h_C);
}
