#include "hip/hip_runtime.h"
/* 
   159735 Parallel Programming

   Startup program for sequential implementation of simulation by ray
   tracing of gravitational lensing.
 */
 #include <ctime>

 #include <iostream>
 #include <string>
 
 #include <cmath>
 
 #include "lenses.h"
 #include "arrayff.hxx"
 #include <hip/hip_runtime.h>
 
 // Global variables! Not nice style, but we'll get away with it here.
 
 // Boundaries in physical units on the lens plane
 const float WL  = 2.0;
 const float XL1 = -WL;
 const float XL2 =  WL;
 const float YL1 = -WL;
 const float YL2 =  WL;
 
 double diffclock(clock_t clock1,clock_t clock2)
{
  double diffticks = clock1 - clock2;
  double diffms = (diffticks * 1000) / CLOCKS_PER_SEC;
  return diffms; // Time difference in milliseconds
}

void lens_demo_seq(int n, float lens_scale) 
{
  // Set up lensing system configuration - call example_1, _2, _3 or
  // _n as you wish. The positions and mass fractions of the lenses
  // are stored in these arrays
  float* xlens;
  float* ylens;
  float* eps;
  if ( n == 1 ) 
    const int nlenses = set_example_1(&xlens, &ylens, &eps);
  else if (n == 2)
    const int nlenses = set_example_2(&xlens, &ylens, &eps);
  else if (n == 3)
    const int nlenses = set_example_3(&xlens, &ylens, &eps);
  else
    const int nlenses = set_example_n(n,&xlens, &ylens, &eps);

  std::cout << "# Simulating " << nlenses << " lens system" << std::endl;

  // Source star parameters. You can adjust these if you like - it is
  // interesting to look at the different lens images that result
  const float rsrc = 0.1;      // radius
  const float ldc  = 0.5;      // limb darkening coefficient
  const float xsrc = 0.0;      // x and y centre on the map
  const float ysrc = 0.0;

  // Pixel size in physical units of the lens image. You can try finer
  // lens scale which will result in larger images (and take more
  // time).
  //const float lens_scale = 0.005;

  // Size of the lens image
  const int npixx = static_cast<int>(floor((XL2 - XL1) / lens_scale)) + 1;
  const int npixy = static_cast<int>(floor((YL2 - YL1) / lens_scale)) + 1;
  std::cout << "# Building " << npixx << "X" << npixy << " lens image" << std::endl;

  // Put the lens image in this array
  Array<float, 2> lensim(npixy, npixx);

  clock_t tstart = clock();

  // Draw the lensing image map here. For each pixel, shoot a ray back
  // to the source plane, then test whether or or not it hits the
  // source star
  const float rsrc2 = rsrc * rsrc;
  float xl, yl, xs, ys, sep2, mu;
  float xd, yd;
  int numuse = 0;
  for (int iy = 0; iy < npixy; ++iy) 
  for (int ix = 0; ix < npixx; ++ix) { 
    
    // YOU NEED TO COMPLETE THIS SECTION OF CODE

    // need position on lens in physical units

    // shoot a ray back to the source plane - make the appropriate
    // call to shoot() in lenses.h

    // does the ray hit the source star?
    shoot(xs, ys, xl, yl, xlens, ylens, eps, nlenses);
    xd = xs - xsrc;
    yd = ys - ysrc;
    sep2 = xd * xd + yd * yd;
    if (sep2 < rsrc2) {
      mu = sqrt(1 - sep2 / rsrc2);
      lensim(iy, ix) = 1.0 - ldc * (1 - mu);
    }
  }

  clock_t tend = clock();
  double tms = diffclock(tend, tstart);
  std::cout << "# Time elapsed in seq: " << tms << " ms " << numuse << std::endl;

  // Write the lens image to a FITS formatted file. You can view this
  // image file using ds9
  //dump_array<float, 2>(lensim, "lens.fit");

  delete[] xlens;
  delete[] ylens;
  delete[] eps;
}
 
 __global__ void lensim_gpu(float* xlens, float* ylens, float* eps, int npixx, int npixy ,int nlenses , float* lensim)
 {
   
   int i = blockDim.x * blockIdx.x + threadIdx.x;
   if ( i >= npixx * npixy) return ;
   const float rsrc = 0.1;      // radius
   const float ldc  = 0.5;      // limb darkening coefficient
   const float xsrc = 0.0;      // x and y centre on the map
   const float ysrc = 0.0;
   const float lens_scale = 0.005;
   const float WL  = 2.0;
   const float XL1 = -WL;
   const float YL1 = -WL;
 
   const float rsrc2 = rsrc * rsrc;
   float xl, yl, xs, ys, sep2, mu;
   float xd, yd;
 
   int iy = i / npixy;
   int ix = i % npixx;
 
   yl = YL1 + iy * lens_scale;
   xl = XL1 + ix * lens_scale;
   //shoot(xs, ys, xl, yl, xlens, ylens, eps, nlenses);
 
   float dx, dy, dr;
   xs = xl;
   ys = yl;
   for (int p = 0; p < nlenses; ++p) {
     dx = xl - xlens[p];
     dy = yl - ylens[p];
     dr = dx * dx + dy * dy;
     xs -= eps[p] * dx / dr;
     ys -= eps[p] * dy / dr;
   }
 
 
   xd = xs - xsrc;
   yd = ys - ysrc;
   sep2 = xd * xd + yd * yd;
   if (sep2 < rsrc2) {
     mu = sqrt(1 - sep2 / rsrc2);
     lensim[i] = 1.0 - ldc * (1 - mu);
   }
 
 }
 
 int main(int argc, char* argv[]) 
 {

  lens_demo_seq(atoi(argv[1]),atof(argv[2])) ;
   // Set up lensing system configuration - call example_1, _2, _3 or
   // _n as you wish. The positions and mass fractions of the lenses
   // are stored in these arrays
   float* xlens;
   float* ylens;
   float* eps;
   //const int nlenses = set_example_n( atoi(argv[1]) ,&xlens, &ylens, &eps);
   const int n = atof(argv[2]);
   if ( n == 1 ) 
    const int nlenses = set_example_1(&xlens, &ylens, &eps);
   else if (n == 2)
    const int nlenses = set_example_2(&xlens, &ylens, &eps);
   else if (n == 3)
    const int nlenses = set_example_3(&xlens, &ylens, &eps);
   else
    const int nlenses = set_example_n(n,&xlens, &ylens, &eps);

   std::cout << "# Simulating " << nlenses << " lens system" << std::endl;
   const float lens_scale = atof(argv[2]) ;
 
   // Size of the lens image
   const int npixx = static_cast<int>(floor((XL2 - XL1) / lens_scale)) + 1;
   const int npixy = static_cast<int>(floor((YL2 - YL1) / lens_scale)) + 1;
   std::cout << "# Building " << npixx << "X" << npixy << " lens image" << std::endl;
   const int npixx_npixy = npixx * npixy;

   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);

   // Put the lens image in this array
   Array<float, 2> lensim(npixy, npixx);
 
   size_t size1 = npixx_npixy * sizeof(float);
   size_t size2 = nlenses * sizeof(float);
   float *d_lensim,*d_xlens,*d_ylens,*d_eps;
 
   hipMalloc(&d_lensim, size1);
   hipMalloc(&d_xlens, size2);
   hipMalloc(&d_ylens, size2);
   hipMalloc(&d_eps, size2);
 
 
   hipMemcpy(d_xlens, xlens, size2, hipMemcpyHostToDevice);
   hipMemcpy(d_ylens, ylens, size2, hipMemcpyHostToDevice);
   hipMemcpy(d_eps, eps, size2, hipMemcpyHostToDevice);
   hipMemcpy(d_lensim, lensim.buffer, size1, hipMemcpyHostToDevice);
 
   int threadsPerBlock = 256;
   int blocksPerGrid = (npixx_npixy + threadsPerBlock - 1) / threadsPerBlock;

   std::cout << "Launching a grid of " 
   << blocksPerGrid << " "
   << threadsPerBlock * blocksPerGrid
   << " threads" << std::endl;

   hipEventRecord(start, 0);
   lensim_gpu<<<blocksPerGrid, threadsPerBlock>>>( d_xlens,  d_ylens,  d_eps,  npixx, npixy , nlenses , d_lensim);

   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   float time;  // Must be a float
   hipEventElapsedTime(&time, start, stop);
   hipEventDestroy(start);
   hipEventDestroy(stop);
   std::cout << "Kernel took: " << time << " ms" << std::endl;

   hipMemcpy( lensim.buffer, d_lensim, size1, hipMemcpyDeviceToHost);
 
   hipFree(d_lensim);
   hipFree(d_xlens);
   hipFree(d_ylens);
   hipFree(d_eps);
 
 
   // Write the lens image to a FITS formatted file. You can view this
   // image file using ds9
   dump_array<float, 2>(lensim, "lens.fit");
 
   delete[] xlens;
   delete[] ylens;
   delete[] eps;
 }
 
   
 